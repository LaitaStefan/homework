#include "hip/hip_runtime.h"
/*
 * Papa Florin, 334CA
 * Tema 4 ASC
 */

/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * This software and the information contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a Non-Disclosure Agreement.  Any reproduction or
 * disclosure to any third party without the express written consent of
 * NVIDIA is prohibited.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>

// includes, project
#include "2Dconvolution.h"


////////////////////////////////////////////////////////////////////////////////
// declarations, forward

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int);

Matrix AllocateDeviceMatrix(int width, int height);
Matrix AllocateMatrix(int width, int height);
void FreeDeviceMatrix(Matrix* M);
void FreeMatrix(Matrix* M);

void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P, FILE *f);
void ConvolutionOnDeviceShared(const Matrix M, const Matrix N, Matrix P, FILE *f);

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{   
    if(row >= 0 && row < A.height && col >= 0 && col < A.width) {
        return A.elements[row * A.width + col];
    }

    return 0;
}

////////////////////////////////////////////////////////////////////////////////
// Înmulțirea fără memorie partajată
////////////////////////////////////////////////////////////////////////////////
__global__ void ConvolutionKernel(Matrix M, Matrix N, Matrix P)
{
    //calcul rezultat convoluție
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int i, j;
    int indexP = row * P.width + col;

    if(row < N.height && col < N.width) {
        P.elements[indexP] = 0;
        for(i = 0; i < M.height; i++) {
            for(j = 0; j < M.width; j++) {
                int rowN = row + i - 2;
                int colN = col + j - 2;

                if(rowN >= 0 && rowN < N.height) {
                    if(colN >= 0 && colN < N.width) {
                        int indexM = i * M.width + j;
                        float Melem = M.elements[indexM];
                        float Nelem = N.elements[rowN * N.width + colN];
                        P.elements[indexP] += Melem * Nelem;
                    }
                }
            }
        }
    }
}


////////////////////////////////////////////////////////////////////////////////
// Înmulțirea cu memorie partajată
////////////////////////////////////////////////////////////////////////////////
__global__ void ConvolutionKernelShared(Matrix M, Matrix N, Matrix P)
{
    int threadCol = blockIdx.x * blockDim.x + threadIdx.x;
    int threadRow = blockIdx.y * blockDim.y + threadIdx.y;
    float Pvalue = 0;
    __shared__ float Ns[SHARED_BLOCK_SIZE][SHARED_BLOCK_SIZE];
    __shared__ float Ms[KERNEL_SIZE][KERNEL_SIZE];
    int row = threadIdx.y;
    int col = threadIdx.x;
    int i, j;

    //calculul rezultatului convoluției

    // Load M and Nsub from device memory to shared memory
    // Each thread loads one element of each sub-matrix
    if(threadCol < N.width && threadRow < N.height) {
        Ns[row + 2][col + 2] = N.elements[threadRow * N.width + threadCol];

        // padding up
        if(row == 0) {
            Ns[row][col + 2] = GetElement(N, threadRow - 2, threadCol);
            Ns[row + 1][col + 2] = GetElement(N, threadRow - 1, threadCol);
        }

        // padding left
        if(col == 0) {
            Ns[row + 2][col] = GetElement(N, threadRow, threadCol - 2);
            Ns[row + 2][col + 1] = GetElement(N, threadRow, threadCol - 1);

            if(row == 0) {
                Ns[row][col] = GetElement(N, threadRow - 2, threadCol - 2);
                Ns[row][col + 1] = GetElement(N, threadRow - 2, threadCol - 1);
                Ns[row + 1][col] = GetElement(N, threadRow - 1, threadCol - 2);
                Ns[row + 1][col + 1] = GetElement(N, threadRow - 1, threadCol - 1);
            }
            else if(row == BLOCK_SIZE - 1 || threadRow == N.height - 1) {
                Ns[row + 3][col] = GetElement(N, threadRow + 1, threadCol - 2);
                Ns[row + 3][col + 1] = GetElement(N, threadRow + 1, threadCol - 1);
                Ns[row + 4][col] = GetElement(N, threadRow + 2, threadCol - 2);
                Ns[row + 4][col + 1] = GetElement(N, threadRow + 2, threadCol - 1);
            }
        }

        //padding down
        if(row == BLOCK_SIZE - 1 || threadRow == N.height - 1) {
            Ns[row + 3][col + 2] = GetElement(N, threadRow + 1, threadCol);
            Ns[row + 4][col + 2] = GetElement(N, threadRow + 2, threadCol);
        }

        // padding right
        if(col == BLOCK_SIZE - 1 || threadCol == N.width - 1) {
            Ns[row + 2][col + 3] = GetElement(N, threadRow, threadCol + 1);
            Ns[row + 2][col + 4] = GetElement(N, threadRow, threadCol + 2);

            if(row == 0) {
                Ns[row][col + 3] = GetElement(N, threadRow - 2, threadCol + 1);
                Ns[row][col + 4] = GetElement(N, threadRow - 2, threadCol + 2);
                Ns[row + 1][col + 3] = GetElement(N, threadRow - 1, threadCol + 1);
                Ns[row + 1][col + 4] = GetElement(N, threadRow - 1, threadCol + 2);
            }
            else if(row == BLOCK_SIZE - 1 || threadRow == N.height - 1) {
                Ns[row + 3][col + 3] = GetElement(N, threadRow + 1, threadCol + 1);
                Ns[row + 3][col + 4] = GetElement(N, threadRow + 1, threadCol + 2);
                Ns[row + 4][col + 3] = GetElement(N, threadRow + 2, threadCol + 1);
                Ns[row + 4][col + 4] = GetElement(N, threadRow + 2, threadCol + 2);
            }
        }
    }

    // load kernel matrix into shared memory
    if(row == 0 && col == 0) {
        for(i = 0; i < KERNEL_SIZE; i++) {
            for(j = 0; j < KERNEL_SIZE; j++) {
                Ms[i][j] = M.elements[i * KERNEL_SIZE + j];
            }
        }
    }

    // Synchronize to make sure the sub-matrices are loaded
    // before starting the computation
    __syncthreads();

    
    if(threadRow < N.height && threadCol < N.width) {
        // perform computations
        for(i = 0; i < KERNEL_SIZE; i++) {
            for(j = 0; j < KERNEL_SIZE; j++) {
                int sharedRow = row + i;
                int sharedCol = col + j;

                Pvalue += Ms[i][j] * Ns[sharedRow][sharedCol];
            }
        }

        P.elements[threadRow * P.width + threadCol] = Pvalue;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Returnează 1 dacă matricele sunt ~ egale
////////////////////////////////////////////////////////////////////////////////
int CompareMatrices(Matrix A, Matrix B)
{
    int i;
    if(A.width != B.width || A.height != B.height || A.pitch != B.pitch)
        return 0;
    int size = A.width * A.height;
    for(i = 0; i < size; i++)
        if(fabs(A.elements[i] - B.elements[i]) > MAX_ERR)
            return 0;
    return 1;
}
void GenerateRandomMatrix(Matrix m)
{
    int i;
    int size = m.width * m.height;

    srand(time(NULL));

    for(i = 0; i < size; i++)
        m.elements[i] = rand() / (float)RAND_MAX;
}

////////////////////////////////////////////////////////////////////////////////
// main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) 
{
    int width = 0, height = 0;
    FILE *f, *out, *times;
    float tm;
    if(argc < 2)
    {
        printf("Argumente prea puține, trimiteți id-ul testului care trebuie rulat\n");
        return 0;
    }
    char name[100];
    sprintf(name, "./tests/test_%s.txt", argv[1]);

    if(atoi(argv[1]) == 1) {
        times = fopen("time_out.txt", "w");
    }
    else {
        times = fopen("time_out.txt", "a");
    }

    f = fopen(name, "r");
    out = fopen("out.txt", "a");
    fscanf(f, "%d%d", &width, &height);

    fprintf(times, "T%s 23 ", argv[1]);
    
    Matrix M;   //kernel de pe host
    Matrix N;   //matrice inițială de pe host
    Matrix P;   //rezultat fără memorie partajată calculat pe GPU
    Matrix PS;  //rezultatul cu memorie partajată calculat pe GPU
	
    M = AllocateMatrix(KERNEL_SIZE, KERNEL_SIZE);
    N = AllocateMatrix(width, height);		
    P = AllocateMatrix(width, height);
    PS = AllocateMatrix(width, height);

    GenerateRandomMatrix(M);
    GenerateRandomMatrix(N);
    
	// M * N pe device
    ConvolutionOnDevice(M, N, P, times);
    
    // M * N pe device cu memorie partajată
    ConvolutionOnDeviceShared(M, N, PS, times);

    // calculează rezultatul pe CPU pentru comparație
    Matrix reference = AllocateMatrix(P.height, P.width);
    
    //pentru măsurarea timpului de execuție în kernel
    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);

    sdkStartTimer(&kernelTime);
    computeGold(reference.elements, M.elements, N.elements, N.height, N.width);
    sdkStopTimer(&kernelTime);
    tm = sdkGetTimerValue(&kernelTime);
    // for plotting purposes
    // 2.5 is a very big time compared to CUDA times
    if(tm > 2.5) tm = 2.5;
    fprintf(times, "%f\n", tm);
        
    // verifică dacă rezultatul obținut pe device este cel așteptat
    int res = CompareMatrices(reference, P);
    printf("Test global %s\n", (1 == res) ? "PASSED" : "FAILED");
    fprintf(stderr, "Test global %s %s\n", argv[1], (1 == res) ? "PASSED" : "FAILED");

    // verifică dacă rezultatul obținut pe device cu memorie partajată este cel așteptat
    int ress = CompareMatrices(reference, PS);
    printf("Test shared %s\n", (1 == ress) ? "PASSED" : "FAILED");
    fprintf(stderr, "Test shared %s %s\n", argv[1], (1 == ress) ? "PASSED" : "FAILED");
   
	// Free matrices
    FreeMatrix(&M);
    FreeMatrix(&N);
    FreeMatrix(&P);
    FreeMatrix(&PS);

    fclose(f);
    fclose(out);
    fclose(times);
    return 0;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void ConvolutionOnDevice(const Matrix M, const Matrix N, Matrix P, FILE *f)
{
    Matrix Md, Nd, Pd; //matricele corespunzătoare de pe device
    int sizeM = M.width * M.height;
    int sizeN = N.width * N.height;
    int sizeP = P.width * P.height;
    float tm;

    //pentru măsurarea timpului de execuție în kernel
    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);
    
    //alocare matrice de pe device
    Md = AllocateDeviceMatrix((int)M.width, (int)M.height);
    Nd = AllocateDeviceMatrix((int)N.width, (int)N.height);
    Pd = AllocateDeviceMatrix((int)P.width, (int)P.height);

    //copiere date de pe host (M, N) pe device (MD, Nd)
    hipMemcpy(Md.elements, M.elements, sizeM * sizeof(float),
                hipMemcpyHostToDevice);
    Md.width = Md.pitch = M.width;
    Md.height = M.height;

    hipMemcpy(Nd.elements, N.elements, sizeN * sizeof(float),
                hipMemcpyHostToDevice);
    Nd.width = Nd.pitch = N.width;
    Nd.height = N.height;

    //setare configurație de rulare a kernelului
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N.width + dimBlock.x - 1) / dimBlock.x,
                (N.height + dimBlock.y - 1) / dimBlock.y);

    sdkStartTimer(&kernelTime);

    //lansare în execuție kernelul
    ConvolutionKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd);

    hipDeviceSynchronize();
    sdkStopTimer(&kernelTime);
    tm = sdkGetTimerValue(&kernelTime);
    fprintf(f, "%f ", tm);
    printf ("Timp execuție kernel: %f ms\n", tm);

    //copiere rezultat in P
    hipMemcpy(P.elements, Pd.elements, sizeP * sizeof(float),
                hipMemcpyDeviceToHost);

    //eliberare memoria matricelor de pe device
    FreeDeviceMatrix(&Md);
    FreeDeviceMatrix(&Nd);
    FreeDeviceMatrix(&Pd);
}


void ConvolutionOnDeviceShared(const Matrix M, const Matrix N, Matrix P, FILE *f)
{
    Matrix Md, Nd, Pd; //matricele corespunzătoare de pe device
    int sizeM = M.width * M.height;
    int sizeN = N.width * N.height;
    int sizeP = P.width * P.height;
    float tm;

    //pentru măsurarea timpului de execuție în kernel
    StopWatchInterface *kernelTime = NULL;
    sdkCreateTimer(&kernelTime);
    sdkResetTimer(&kernelTime);

    //alocare matricele de pe device
    Md = AllocateDeviceMatrix((int)M.width, (int)M.height);
    Nd = AllocateDeviceMatrix((int)N.width, (int)N.height);
    Pd = AllocateDeviceMatrix((int)P.width, (int)P.height);

    //copiere date de pe host (M, N) pe device (MD, Nd)
    hipMemcpy(Md.elements, M.elements, sizeM * sizeof(float), hipMemcpyHostToDevice);
    Md.width = Md.pitch = M.width;
    Md.height = M.height;

    hipMemcpy(Nd.elements, N.elements, sizeN * sizeof(float), hipMemcpyHostToDevice);
    Nd.width = Nd.pitch = N.width;
    Nd.height = N.height;

    //setare configurație de rulare a kernelului
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N.width + dimBlock.x - 1) / dimBlock.x,
                (N.height + dimBlock.y - 1) / dimBlock.y);

    sdkStartTimer(&kernelTime);

    //lansare în execuție kernel
    ConvolutionKernelShared<<<dimGrid, dimBlock>>>(Md, Nd, Pd);

    hipDeviceSynchronize();
    sdkStopTimer(&kernelTime);
    tm = sdkGetTimerValue(&kernelTime);
    fprintf(f, "%f ", tm);
    printf ("Timp execuție kernel cu memorie partajată: %f ms\n", tm);

    //copiere rezultat in P
    hipMemcpy(P.elements, Pd.elements, sizeP * sizeof(float),
                hipMemcpyDeviceToHost);

    //eliberare memoria matricelor de pe device
    FreeDeviceMatrix(&Md);
    FreeDeviceMatrix(&Nd);
    FreeDeviceMatrix(&Pd);
}


// Alocă o matrice de dimensiune height*width pe device
Matrix AllocateDeviceMatrix(int width, int height)
{
    Matrix m;
    int size;

    m.width = m.pitch = width;
    m.height = height;
    size = width * height;

    hipMalloc( (void**) &(m.elements), size *sizeof(float));

    return m;
}

// Alocă matrice pe host de dimensiune height*width
Matrix AllocateMatrix(int height, int width)
{
    Matrix M;
    M.width = M.pitch = width;
    M.height = height;
    int size = M.width * M.height;
	M.elements = (float*) malloc(size*sizeof(float));
    return M;
}	

// Eliberează o matrice de pe device
void FreeDeviceMatrix(Matrix* M)
{
    hipFree(M->elements);
    M->elements = NULL;
}

// Eliberează o matrice de pe host
void FreeMatrix(Matrix* M)
{
    free(M->elements);
    M->elements = NULL;
}
